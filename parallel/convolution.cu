#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>

#include "convolution.h"

#define TILE_WIDTH 32
#define MAX_MASK_WIDTH 10

#define clamp(start, x, end) (fmin(fmax(start, x), end))
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__, __LINE__, #value, value)


// Check the return value of the CUDA runtime API call and exit the application if the call has failed.
static void CheckCudaErrorAux(const char *file, unsigned line, const char *statement, hipError_t err) {
    if (err == hipSuccess){
        return;
    }
    std::cerr << statement << " returned " << hipGetErrorString(err) << "(" << err << ") at " << file << ":" << line << std::endl;
    exit(1);
}


// Get the pixel value at the specified coordinates.
__device__ uint8_t& get_pixel_value(uint8_t* d_input, const int col, const int row, const int channel, const int width, const int height, const int channels, const bool is_SoA) {
    // Get the 1D pixel index.
    const int pixel_index = is_SoA ? ((channel * width * height) + (row * width) + col) : ((row * width + col) * channels + channel);

    return d_input[pixel_index];
}

// Set the pixel value at the specified coordinates.
__device__ void set_pixel_value(uint8_t* d_input, const int col, const int row, const int channel, const int width, const int height, const int channels, const bool is_SoA, const uint8_t value) {
    // Get the 1D pixel index.
    const int pixel_index = is_SoA ? ((channel * width * height) + (row * width) + col) : ((row * width + col) * channels + channel);

    d_input[pixel_index] = (uint8_t)value;
}

// Get the kernel value at the specified coordinates.
__device__ float& get_kernel_value(float* d_kernel, const int col, const int row, const int width, const int height) {
    // Get the 1D kernel index.
    const int kernel_index = (row * width) + col;

    return d_kernel[kernel_index];
}

// Get the pixel value at the specified coordinates from shared memory.
__device__ uint8_t& get_shared_pixel_value(uint8_t* s_data, const int col, const int row, const int width, const int height) {
    // Get the 1D pixel index.
    const int pixel_index = (row * width) + col;

    return s_data[pixel_index];
}


// Kernel function for convolution using global memory.
__global__ void convolution_kernel_global(uint8_t* d_input, float* d_kernel, uint8_t* d_output,
                                   int width, int height, int channels,
                                   int kernel_width, int kernel_height,
                                   int padding_width, int padding_height, bool is_SoA)
{
    // Calculate the global index in the output image.
    const int x = blockIdx.x * blockDim.x + threadIdx.x; // Column index.
    const int y = blockIdx.y * blockDim.y + threadIdx.y; // Row index.
    const int channel = blockIdx.z * blockDim.z + threadIdx.z; // Channel index.

    // Padded image dimensions.
    const int padded_width = width + (2 * padding_width); // Padded image width.
    const int padded_height = height + (2 * padding_height); // Padded image height.


    // Check if the thread is within the image bounds.
    if(x < width && y < height && channel < channels) {
        // Output value for the current pixel.
        float output_value = 0.0f;

        // Iterate over the kernel.
        for(int ky = 0; ky < kernel_height; ky++) {
            for(int kx = 0; kx < kernel_width; kx++) {
                // Get the pixel index to be convolved.
                const int col = x + kx - floor((float)kernel_width / 2) + padding_width;
                const int row = y + ky - floor((float)kernel_height / 2) + padding_height; 

                // Add the convolution value to the output value.
                output_value += get_pixel_value(d_input, col, row, channel, padded_width, padded_height, channels, is_SoA) * get_kernel_value(d_kernel, kx, ky, kernel_width, kernel_height);
            }
        }

        // Store the output value in global memory.
        set_pixel_value(d_output, x, y, channel, width, height, channels, is_SoA, (uint8_t)clamp(0.0f, output_value, 255.0f));
    }
}


// Constant memory to store the kernel.
__constant__ float c_kernel[MAX_MASK_WIDTH * MAX_MASK_WIDTH];

// Kernel function for convolution using constant memory.
__global__ void convolution_kernel_constant(uint8_t* d_input, uint8_t* d_output,
                                   int width, int height, int channels,
                                   int kernel_width, int kernel_height,
                                   int padding_width, int padding_height, bool is_SoA)
{
    // Calculate the global index in the output image.
    const int x = blockIdx.x * blockDim.x + threadIdx.x; // Column index.
    const int y = blockIdx.y * blockDim.y + threadIdx.y; // Row index.
    const int channel = blockIdx.z * blockDim.z + threadIdx.z; // Channel index.

    // Padded image dimensions.
    const int padded_width = width + (2 * padding_width); // Padded image width.
    const int padded_height = height + (2 * padding_height); // Padded image height.


    // Check if the thread is within the image bounds.
    if(x < width && y < height && channel < channels) {
        // Output value for the current pixel.
        float output_value = 0.0f;

        // Iterate over the kernel.
        for(int ky = 0; ky < kernel_height; ky++) {
            for(int kx = 0; kx < kernel_width; kx++) {
                // Get the pixel index to be convolved.
                const int col = x + kx - floor((float)kernel_width / 2) + padding_width;
                const int row = y + ky - floor((float)kernel_height / 2) + padding_height;

                // Add the convolution value to the output value.
                output_value += get_pixel_value(d_input, col, row, channel, padded_width, padded_height, channels, is_SoA) * get_kernel_value(c_kernel, kx, ky, kernel_width, kernel_height);
            }
        }

        // Store the output value in global memory.
        set_pixel_value(d_output, x, y, channel, width, height, channels, is_SoA, (uint8_t)clamp(0.0f, output_value, 255.0f));
    }

}


// Kernel function for convolution using shared memory.
__global__ void convolution_kernel_shared(uint8_t* d_input, uint8_t* d_output,
                                   int width, int height, int channels,
                                   int kernel_width, int kernel_height,
                                   int padding_width, int padding_height, bool is_SoA)
{
    // Shared memory for the input image tile (dynamically sized by kernel launcher).
	extern __shared__ uint8_t s_data[];

    // Shared memory dimensions.
    const int s_width = blockDim.x + (kernel_width - 1); // Shared memory width.
    const int s_height = blockDim.y + (kernel_height - 1); // Shared memory height.

    // Padded image dimensions.
    const int padded_width = width + (2 * padding_width); // Padded image width.
    const int padded_height = height + (2 * padding_height); // Padded image height.


    // Loading first (blockDim.x * blockDim.y) elements into shared memory.

    // Shared memory index.
    int s_index = (threadIdx.y * blockDim.x) + threadIdx.x; // Shared memory index.
    int s_x = s_index % s_width; // Shared memory column index.
    int s_y = s_index / s_width; // Shared memory row index.

    // Global memory index to load the input image tile.
    int x = (blockIdx.x * blockDim.x) + s_x - floor((float) kernel_width / 2); // Global memory column index.
    int y = (blockIdx.y * blockDim.y) + s_y - floor((float) kernel_height / 2); // Global memory row index.
    int channel = blockIdx.z * blockDim.z + threadIdx.z; // Global memory channel index.

    // Check if the thread is within the image bounds.
    if(x >= 0 && x < padded_width && y >= 0 && y < padded_height) {
        // Load the pixel value into shared memory.
        s_data[s_index] = get_pixel_value(d_input, x, y, channel, padded_width, padded_height, channels, is_SoA);
    } else {
        // Load 0 into shared memory.
        s_data[s_index] = 0;
    }
    
    
    // Loading last (s_width * s_height) - (blockDim.x * blockDim.y) elements into shared memory.

    // Shared memory index.
    s_index = (threadIdx.y * blockDim.x) + threadIdx.x + (blockDim.x * blockDim.y); // Shared memory index.
    s_x = s_index % s_width; // Shared memory column index.
    s_y = s_index / s_width; // Shared memory row index.

    // Global memory index to load the input image tile.
    x = (blockIdx.x * blockDim.x) + s_x - floor((float) kernel_width / 2); // Global memory column index.
    y = (blockIdx.y * blockDim.y) + s_y - floor((float) kernel_height / 2); // Global memory row index.
    channel = blockIdx.z * blockDim.z + threadIdx.z; // Global memory channel index.

    // Check if the thread is within the image bounds.
    if(s_y < s_height) {
        if(x >= 0 && x < padded_width && y >= 0 && y < padded_height) {
            // Load the pixel value into shared memory.
            s_data[s_index] = get_pixel_value(d_input, x, y, channel, padded_width, padded_height, channels, is_SoA);
        } else {
            // Load 0 into shared memory.
            s_data[s_index] = 0;
        }
    }

    // Wait for all threads to finish loading the input image tile into shared memory.
    __syncthreads();

    
    // Convolve the input image tile with the kernel.

    // Output value for the current pixel.
    float output_value = 0.0f;

    // Iterate over the kernel.
    for(int ky = 0; ky < kernel_height; ky++) {
        for(int kx = 0; kx < kernel_width; kx++) {
            // Get the pixel index to be convolved.
            const int col = threadIdx.x + kx;
            const int row = threadIdx.y + ky;

            // Add the convolution value to the output value.
            output_value += get_shared_pixel_value(s_data, col, row, s_width, s_height) * get_kernel_value(c_kernel, kx, ky, kernel_width, kernel_height);
        }
    }


    // Calculate the global index in the output image.
    x = blockIdx.x * blockDim.x + threadIdx.x - padding_width;
    y = blockIdx.y * blockDim.y + threadIdx.y - padding_height;
    channel = blockIdx.z * blockDim.z + threadIdx.z;

    // Store the output value in global memory.
    if (x >= 0 && x < width && y >= 0 && y < height && channel < channels) {
        set_pixel_value(d_output, x, y, channel, width, height, channels, is_SoA, (uint8_t)clamp(0.0f, output_value, 255.0f));
    }

    // Wait for all threads to finish convolving the input image tile with the kernel.
    //__syncthreads();
}


// Methods.

Image Parallel::Convolution::convolve_global(const Image& image, const Kernel& kernel, PaddingType padding_type) {
    // Input image dimensions.
    const int width = image.get_width(); // Input image width.
    const int height = image.get_height(); // Input image height.
    const int channels = image.get_channels(); // Input image channels.

    // Kernel dimensions.
    const int kernel_width = kernel.get_width(); // Kernel width.
    const int kernel_height = kernel.get_height(); // Kernel height.


    // Apply padding to the input image.
    const int padding_width = floor((float)kernel_width / 2); // Padding width.
    const int padding_height = floor((float)kernel_height / 2); // Padding height.
    Image padded_image = image.padding(padding_width, padding_height, padding_type); // Padded image.

    // Padded image dimensions.
    const int padded_width = padded_image.get_width(); // Padded image width.
    const int padded_height = padded_image.get_height(); // Padded image height.
    const int padded_channels = padded_image.get_channels(); // Padded image channels.


    // Sizes in bytes.
    const int input_size = padded_width * padded_height * padded_channels * sizeof(uint8_t); // Input image size.
    const int output_size = width * height * channels * sizeof(uint8_t); // Output image size.
    const int kernel_size = kernel_width * kernel_height * sizeof(float); // Kernel size.
    

    // Host memory pointers.
    uint8_t* h_input = padded_image.get_data(); // Input image data.
    uint8_t* h_output = (uint8_t*)malloc(output_size); // Output image data.
    float* h_kernel = kernel.get_data(); // Kernel data.

    // Device memory pointers.
    uint8_t* d_input; // Input image data.
    uint8_t* d_output; // Output image data.
    float* d_kernel; // Kernel data.


    // Allocate device memory.
    CUDA_CHECK_RETURN(hipMalloc((void**)&d_input, input_size));
    CUDA_CHECK_RETURN(hipMalloc((void**)&d_output, output_size));
    CUDA_CHECK_RETURN(hipMalloc((void**)&d_kernel, kernel_size));

    // Copy data from host to device global memory.
    CUDA_CHECK_RETURN(hipMemcpy(d_input, h_input, input_size, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(d_kernel, h_kernel, kernel_size, hipMemcpyHostToDevice));


    // Specify block and grid dimensions.
    dim3 blockDim(TILE_WIDTH, TILE_WIDTH, 1); // Threads per block.
    dim3 gridDim(ceil((float)width / TILE_WIDTH), ceil((float)height / TILE_WIDTH), channels); // Blocks per grid.

    // Launch kernel.
    convolution_kernel_global<<<gridDim, blockDim>>>(d_input, d_kernel, d_output, width, height, channels, kernel_width, kernel_height, padding_width, padding_height, image.get_is_SoA());

    // Waits for threads to finish work.
	hipDeviceSynchronize();

    // Copy output data from device global memory to host memory.
    CUDA_CHECK_RETURN(hipMemcpy(h_output, d_output, output_size, hipMemcpyDeviceToHost));

    // Clean up device memory after kernel execution.
    CUDA_CHECK_RETURN(hipFree(d_input));
    CUDA_CHECK_RETURN(hipFree(d_output));
    CUDA_CHECK_RETURN(hipFree(d_kernel));

    // Create the output image.
    return Image(width, height, channels, h_output, image.get_is_SoA());
}

Image Parallel::Convolution::convolve_constant(const Image &image, const Kernel &kernel, PaddingType padding_type) {
    // Input image dimensions.
    const int width = image.get_width(); // Input image width.
    const int height = image.get_height(); // Input image height.
    const int channels = image.get_channels(); // Input image channels.

    // Kernel dimensions.
    const int kernel_width = kernel.get_width(); // Kernel width.
    const int kernel_height = kernel.get_height(); // Kernel height.


    // Apply padding to the input image.
    const int padding_width = floor((float)kernel_width / 2); // Padding width.
    const int padding_height = floor((float)kernel_height / 2); // Padding height.
    Image padded_image = image.padding(padding_width, padding_height, padding_type); // Padded image.

    // Padded image dimensions.
    const int padded_width = padded_image.get_width(); // Padded image width.
    const int padded_height = padded_image.get_height(); // Padded image height.
    const int padded_channels = padded_image.get_channels(); // Padded image channels.


    // Sizes in bytes.
    const int input_size = padded_width * padded_height * padded_channels * sizeof(uint8_t); // Input image size.
    const int output_size = width * height * channels * sizeof(uint8_t); // Output image size.
    const int kernel_size = kernel_width * kernel_height * sizeof(float); // Kernel size.

    
    // Host memory pointers.
    uint8_t* h_input = padded_image.get_data(); // Input image data.
    uint8_t* h_output = (uint8_t*)malloc(output_size); // Output image data.
    float* h_kernel = kernel.get_data(); // Kernel data.

    // Device memory pointers.
    uint8_t* d_input; // Input image data.
    uint8_t* d_output; // Output image data.


    // Allocate device memory.
    CUDA_CHECK_RETURN(hipMalloc((void**)&d_input, input_size));
    CUDA_CHECK_RETURN(hipMalloc((void**)&d_output, output_size));

    // Copy input data from host to device global memory.
    CUDA_CHECK_RETURN(hipMemcpy(d_input, h_input, input_size, hipMemcpyHostToDevice));

    // Copy kernel data from host to device constant memory.
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(c_kernel), h_kernel, kernel_size, 0, hipMemcpyHostToDevice));


    // Specify block and grid dimensions.
    dim3 blockDim(TILE_WIDTH, TILE_WIDTH, 1); // Threads per block.
    dim3 gridDim(ceil((float)width / TILE_WIDTH), ceil((float)height / TILE_WIDTH), channels); // Blocks per grid.

    // Launch kernel.
    convolution_kernel_constant<<<gridDim, blockDim>>>(d_input, d_output, width, height, channels, kernel_width, kernel_height, padding_width, padding_height, image.get_is_SoA());

    // Waits for threads to finish work.
    hipDeviceSynchronize();

    // Copy output data from device global memory to host memory.
    CUDA_CHECK_RETURN(hipMemcpy(h_output, d_output, output_size, hipMemcpyDeviceToHost));

    // Clean up device memory after kernel execution.
    CUDA_CHECK_RETURN(hipFree(d_input));
    CUDA_CHECK_RETURN(hipFree(d_output));

    // Create the output image.
    return Image(width, height, channels, h_output, image.get_is_SoA());
}

Image Parallel::Convolution::convolve_shared(const Image &image, const Kernel &kernel, PaddingType padding_type) {
    // Input image dimensions.
    const int width = image.get_width(); // Input image width.
    const int height = image.get_height(); // Input image height.
    const int channels = image.get_channels(); // Input image channels.

    // Kernel dimensions.
    const int kernel_width = kernel.get_width(); // Kernel width.
    const int kernel_height = kernel.get_height(); // Kernel height.


    // Apply padding to the input image.
    const int padding_width = floor((float)kernel_width / 2); // Padding width.
    const int padding_height = floor((float)kernel_height / 2); // Padding height.
    Image padded_image = image.padding(padding_width, padding_height, padding_type); // Padded image.

    // Padded image dimensions.
    const int padded_width = padded_image.get_width(); // Padded image width.
    const int padded_height = padded_image.get_height(); // Padded image height.
    const int padded_channels = padded_image.get_channels(); // Padded image channels.


    // Sizes in bytes.
    const int input_size = padded_width * padded_height * padded_channels * sizeof(uint8_t); // Input image size.
    const int output_size = width * height * channels * sizeof(uint8_t); // Output image size.
    const int kernel_size = kernel_width * kernel_height * sizeof(float); // Kernel size.
    const int shared_size = (TILE_WIDTH + kernel_width - 1) * (TILE_WIDTH + kernel_height - 1) * sizeof(uint8_t); // Shared memory size.
    
    
    // Host memory pointers.
    uint8_t* h_input = padded_image.get_data(); // Input image data.
    uint8_t* h_output = (uint8_t*)malloc(output_size); // Output image data.
    float* h_kernel = kernel.get_data(); // Kernel data.

    // Device memory pointers.
    uint8_t* d_input; // Input image data.
    uint8_t* d_output; // Output image data.


    // Allocate device memory.
    CUDA_CHECK_RETURN(hipMalloc((void**)&d_input, input_size));
    CUDA_CHECK_RETURN(hipMalloc((void**)&d_output, output_size));

    // Copy input data from host to device global memory.
    CUDA_CHECK_RETURN(hipMemcpy(d_input, h_input, input_size, hipMemcpyHostToDevice));

    // Copy kernel data from host to device constant memory.
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(c_kernel), h_kernel, kernel_size, 0, hipMemcpyHostToDevice));


    // Specify block and grid dimensions.
    dim3 blockDim(TILE_WIDTH, TILE_WIDTH, 1); // Threads per block.
    dim3 gridDim(ceil((float)width / TILE_WIDTH), ceil((float)height / TILE_WIDTH), channels); // Blocks per grid.

    // Launch kernel.
    convolution_kernel_shared<<<gridDim, blockDim, shared_size>>>(d_input, d_output, width, height, channels, kernel_width, kernel_height, padding_width, padding_height, image.get_is_SoA());

    // Waits for threads to finish work.
    hipDeviceSynchronize();
    
    // Copy output data from device global memory to host memory.
    CUDA_CHECK_RETURN(hipMemcpy(h_output, d_output, output_size, hipMemcpyDeviceToHost));

    // Clean up device memory after kernel execution.
    CUDA_CHECK_RETURN(hipFree(d_input));
    CUDA_CHECK_RETURN(hipFree(d_output));

    // Create the output image.
    return Image(width, height, channels, h_output, image.get_is_SoA());
}
